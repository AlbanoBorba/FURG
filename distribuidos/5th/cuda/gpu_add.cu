
#include <hip/hip_runtime.h>
#include <iostream>
#include <math.h>

// Kernel Function to tell GPU that it can run there
__global__
void add(int n, float *x, float *y){
    int index = threadIdx.x;
    int stride = blockDim.x;
    for (int i=index; i<n; i+=stride)
        y[i] = x[i] + y[i];
}

int main(void){
    int N = 1<<20; // 1M elements
    
    // allocate unified memory: accessible from CPU or GPU
    float *x, *y;
    hipMallocManaged(&x, N*sizeof(float));
    hipMallocManaged(&y, N*sizeof(float));

    for (int i=0; i<N; i++){
        x[i] = 1.0f;
        y[i] = 2.0f;
    }

    // run kernel function on 1M elements on the GPU
    add<<<1, 256>>>(N, x, y);

    // wait for GPU to finish
    hipDeviceSynchronize();

    float maxError = 0.0f;
    for (int i=0; i<N; i++)
        maxError = fmax(maxError, fabs(y[i]-3.0f));
    std::cout << "Max error: " << maxError << std::endl;

    // free memory
    hipFree(x);
    hipFree(y);
    
    return 0;
}