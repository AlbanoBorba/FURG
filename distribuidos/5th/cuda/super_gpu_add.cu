
#include <hip/hip_runtime.h>
#include <iostream>
#include <math.h>

// Kernel Function to tell GPU that it can run there
__global__
void add(int n, float *x, float *y){
    int index = blockIdx.x * blockDim.x + threadIdx.x;
    int stride = blockDim.x * gridDim.x;
    for (int i=index; i<n; i+=stride)
        y[i] = x[i] + y[i];
}

int main(void){
    int N = 1<<20; // 1M elements
    int blockSize = 256;
    int numBlocks = (N + blockSize - 1) / blockSize;
    
    // allocate unified memory: accessible from CPU or GPU
    float *x, *y;
    hipMallocManaged(&x, N*sizeof(float));
    hipMallocManaged(&y, N*sizeof(float));

    for (int i=0; i<N; i++){
        x[i] = 1.0f;
        y[i] = 2.0f;
    }

    // run kernel function on 1M elements on the GPU
    add<<<numBlocks, blockSize>>>(N, x, y);

    // wait for GPU to finish
    hipDeviceSynchronize();

    float maxError = 0.0f;
    for (int i=0; i<N; i++)
        maxError = fmax(maxError, fabs(y[i]-3.0f));
    std::cout << "Max error: " << maxError << std::endl;

    // free memory
    hipFree(x);
    hipFree(y);
    
    return 0;
}